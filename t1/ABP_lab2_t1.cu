

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>

#define num_thread 512
#define warp 32


__global__ void result(float *y,
                       uint count
                           )
{
    for(int i=0;i<count;i++){
        printf("i=%i, %f\n",i, y[i]);
    }
    
}




__global__ void compute(uint   M,
                        uint   N,
                        float *A,
                        float *x,
                        float *y)
{
    __shared__ float sx[warp];
    __shared__ float sy[warp];
    if(threadIdx.x<32 && threadIdx.y==0){
        sy[threadIdx.x]=0;
    }
               float temp=0;

    const int mat_x = blockIdx.x*blockDim.x + threadIdx.y;
    const int mat_y = blockIdx.y*blockDim.y + threadIdx.x;
    const int mat_idx = mat_x*(int)M + mat_y;

    if(threadIdx.x<32 && threadIdx.y==0 && mat_x<(int)N && mat_y<(int)M){
        sx[threadIdx.x] = x[mat_y];
    }
    __syncthreads();

    if(mat_x<(int)N && mat_y<(int)M){
        temp = A[mat_idx] * sx[threadIdx.y];
        atomicAdd(&sy[threadIdx.x], temp);
    }
    __syncthreads();

    if(threadIdx.x<32 && threadIdx.y==0){
        atomicAdd(&y[mat_y], sy[threadIdx.x]);
    }
    __syncthreads();
}




__global__ void setmemoryf(float *A,
                           float value, 
                           size_t count)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<count){
        A[idx] = value;
    }
    // if(threadIdx.x==0 && blockIdx.x==0){
    // for(int i=0;i<count;i++){
    //     printf("i=%i, %f\n",i, A[i]);
    // }}
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    // const int num_SM = 32;
    // const int num_warp = 32;
    // 4 warp schedulers.

    // input parameter
    uint M = 102;
    uint N = 1025;
    //size_t num_thread = 512;
    dim3 threadblock(warp, warp);
    dim3 blockgrid( (uint)(ceilf((float)N / (float)warp) ), (uint)(ceilf((float)M / (float)warp) ) );
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, M * sizeof(float));

    // initialize variable value
    //cudaMemset is only for integer!!!
    //cudaMemset( A, 1, M*N);
    //cudaMemset( x, 3, N);
    //cudaMemset( y, 5, M);
    setmemoryf<<<(uint)(ceilf((float)M*N / (float)num_thread) ), num_thread, 0, 0>>>(A, 1.0f, M*N);
    setmemoryf<<<(uint)(ceilf((float)N / (float)num_thread) ), num_thread, 0, 0>>>(x, 3.0f, N);
    setmemoryf<<<(uint)(ceilf((float)M / (float)num_thread) ), num_thread, 0, 0>>>(y, 0.0f, M);
    hipDeviceSynchronize();

    // compute and timing
    struct timespec start, end;
    timespec_get(&start, TIME_UTC);

    int repeat = 1;
    for(int i=0;i<repeat;i++){
        compute<<<blockgrid, threadblock, 0, 0>>>(M, N, A, x, y );
        hipDeviceSynchronize();
    }

    timespec_get(&end, TIME_UTC);
    time_t d_sec  = end.tv_sec  - start.tv_sec;
    long   d_nsec = end.tv_nsec - start.tv_nsec;
    double total_time = (double)d_sec + (double)d_nsec/1000000000.0;
    printf("time=%f\n", total_time);
    double perf = 1.0e-9 * (double)(M*N + M + N) * 4.0 / total_time * (double)repeat;
    printf("memory throughput = %f GByte/s\n", perf);

    // show result
    result<<<1,1>>>(y,M);


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}