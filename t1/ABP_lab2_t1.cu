#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>


__global__ void compute(const int   M,
                        const int   N,
                        const float *x,
                        const float *y,
                        const float *z)
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        z[idx] = a * x[idx] + y[idx];
    }
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    const int num_SM = 32;
    const int num_warp = 32;

    // input parameter
    int M = 1024;
    int N = 512;
    int num_thread = 1024;
    int num_block = (int)(ceilf((float)M*N / (float)num_thread))
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, N * sizeof(float));

    // initialize variable value
    hipMemset( A, 1, M*N);
    hipMemset( x, 3, N);
    hipMemset( y, 5, N);
    hipDeviceSynchronize();

    // compute
    compute<<<num_block, num_thread>>>()


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}