#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>

 #define num_thread 512


__device__ float reduction(float *sdata
                           )
{
    float sum = 0;
    //atomicAdd(&sum, sdata[threadIdx.x]);
    __syncthreads();
    return sum;
}




__global__ void compute(uint   M,
                        uint   N,
                        float *A,
                        float *x,
                        float *y)
{
    extern __shared__ float sdata[];
    const int idx = blockIdx.y * (int)N + blockIdx.x * blockDim.x + threadIdx.x;
    const int idxx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idxx < N){
        sdata[threadIdx.x] = A[idx] * x[idxx];
    }
    __syncthreads();

    atomicAdd(&sdata[num_thread], sdata[threadIdx.x]);
    __syncthreads();
    if(threadIdx.x == 0){
        atomicAdd(&y[blockIdx.y], sdata[num_thread]);
    }
}




__global__ void setmemoryf(float *A,
                           float value, 
                           size_t count)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<count){
        A[idx] = value;
    }
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    //const int num_SM = 32;
    //const int num_warp = 32;

    // input parameter
    uint M = 1024;
    uint N = 1024;
    //size_t num_thread = 512;
    dim3 grid = ( (uint)(ceilf((float)N / (float)num_thread) ), M );
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, M * sizeof(float));

    // initialize variable value
    //hipMemset is only for integer!!!
    //hipMemset( A, 1, M*N);
    //hipMemset( x, 3, N);
    //hipMemset( y, 5, M);
    setmemoryf<<<(uint)(ceilf((float)M*N / (float)num_thread) ), num_thread, 0, 0>>>(A, 1.0f, M*N);
    setmemoryf<<<(uint)(ceilf((float)N / (float)num_thread) ), num_thread, 0, 0>>>(x, 3.0f, N);
    setmemoryf<<<(uint)(ceilf((float)M / (float)num_thread) ), num_thread, 0, 0>>>(y, 5.0f, M);
    hipDeviceSynchronize();

    // compute
    compute<<<grid, num_thread, (num_thread+1)*sizeof(float), 0>>>(M, N, A, x, y );
    hipDeviceSynchronize();

    // show result
    result<<<1,1>>>();


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}