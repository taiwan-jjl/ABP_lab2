

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>

#define num_thread 512


__global__ void result(float *y,
                       uint count
                           )
{
    for(int i=0;i<count;i++){
        printf("i=%i, %f\n",i, y[i]);
    }
    
}




__global__ void compute(uint   M,
                        uint   N,
                        float *A,
                        float *x,
                        float *y)
{
    extern __shared__ float sdata[];
    const int idx = blockIdx.y * (int)N + blockIdx.x * blockDim.x + threadIdx.x;
    const int idxx = threadIdx.x + blockIdx.x * blockDim.x;

    if(idxx < N){
        sdata[threadIdx.x] = A[idx] * x[idxx];
    }
    __syncthreads();

    if(threadIdx.x != 0){
        atomicAdd(&sdata[0], sdata[threadIdx.x]);
    }
    __syncthreads();
    if(threadIdx.x == 0){
        atomicAdd(&y[blockIdx.y], sdata[0]);
    }
}




__global__ void setmemoryf(float *A,
                           float value, 
                           size_t count)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<count){
        A[idx] = value;
    }
    // if(threadIdx.x==0 && blockIdx.x==0){
    // for(int i=0;i<count;i++){
    //     printf("i=%i, %f\n",i, A[i]);
    // }}
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    //const int num_SM = 32;
    //const int num_warp = 32;
    // 4 warp schedulers.

    // input parameter
    uint M = 1024;
    uint N = 1024;
    //size_t num_thread = 512;
    dim3 grid( (uint)(ceilf((float)N / (float)num_thread) ), M );
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, M * sizeof(float));

    // initialize variable value
    //cudaMemset is only for integer!!!
    //cudaMemset( A, 1, M*N);
    //cudaMemset( x, 3, N);
    //cudaMemset( y, 5, M);
    setmemoryf<<<(uint)(ceilf((float)M*N / (float)num_thread) ), num_thread, 0, 0>>>(A, 1.0f, M*N);
    setmemoryf<<<(uint)(ceilf((float)N / (float)num_thread) ), num_thread, 0, 0>>>(x, 3.0f, N);
    setmemoryf<<<(uint)(ceilf((float)M / (float)num_thread) ), num_thread, 0, 0>>>(y, 0.0f, M);
    hipDeviceSynchronize();

    // compute and timing
    struct timespec start, end;
    timespec_get(&start, TIME_UTC);

    int repeat = 100000;
    for(int i=0;i<repeat;i++){
        compute<<<grid, num_thread, num_thread*sizeof(float), 0>>>(M, N, A, x, y );
        hipDeviceSynchronize();
    }

    timespec_get(&end, TIME_UTC);
    time_t d_sec  = end.tv_sec  - start.tv_sec;
    long   d_nsec = end.tv_nsec - start.tv_nsec;
    double total_time = (double)d_sec + (double)d_nsec/1000000000.0;
    printf("time=%f\n", total_time);
    double perf = 

    // show result
    //result<<<1,1>>>(y,M);


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}