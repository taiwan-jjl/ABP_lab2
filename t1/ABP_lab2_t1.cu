#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>


__device__ float reduction(float *sdata
                           )
{
    float sum = 0;
    atomicAdd(&sum, sdata);
    __syncthreads();
    return sum;
}




__global__ void compute(uint   M,
                        uint   N,
                        float *x,
                        float *y)
{
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ float sdata[];

    if(idx < N){
        sdata[threadIdx.x] = x[idx] * y[idx];
    }
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    //const int num_SM = 32;
    //const int num_warp = 32;

    // input parameter
    uint M = 1024;
    uint N = 512;
    size_t num_thread = 512;
    dim3 num_block = ( (uint)(ceilf((float)N / (float)num_thread) ), M );
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, N * sizeof(float));

    // initialize variable value
    hipMemset( A, 1, M*N);
    hipMemset( x, 3, N);
    hipMemset( y, 5, N);
    hipDeviceSynchronize();

    // compute
    compute<<<num_block, num_thread, num_thread*sizeof(float), 0>>>(M, N, x, y );


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);

    return 0;
}