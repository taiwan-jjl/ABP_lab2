

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <inttypes.h>
#include <stddef.h>
#include <stdbool.h>
#include <time.h>
#include "hipblas.h"

#define num_thread 512
#define warp 32


__global__ void result(float *y,
                       int count
                           )
{
    for(int i=0;i<count;i++){
        printf("i=%i, %f\n",i, y[i]);
    }
    
}




__global__ void compute(int    M,
                        int    N,
                        float *A,
                        float *x,
                        float *y)
{
    __shared__ float sx[warp];
    __shared__ float sy[warp];
    if(threadIdx.y==0){
        sy[threadIdx.x]=0.0f;
    }
               float temp=0.0f;

    const int mat_x = blockIdx.y*blockDim.y + threadIdx.y;
    const int mat_y = blockIdx.x*blockDim.x + threadIdx.x;
    const int mat_idx = mat_x*M + mat_y;

    if(threadIdx.x==0 && mat_x<N){
        sx[threadIdx.y] = x[mat_x];
    }
    __syncthreads();

    if(mat_x<N && mat_y<M){
        temp = A[mat_idx] * sx[threadIdx.y];
        atomicAdd(&sy[threadIdx.x], temp);
    }
    __syncthreads();

    if(threadIdx.y==0 && mat_x<N && mat_y<M){
        atomicAdd(&y[mat_y], sy[threadIdx.x]);
    }
    __syncthreads();

    // if(threadIdx.x==0 && threadIdx.y==0){
    //     printf("blk-x=%u, blk-y=%u \n", blockIdx.x, blockIdx.y);
    // }
}




__global__ void setmemoryf(float *A,
                           float value, 
                           int   count)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<count){
        A[idx] = value;
    }
    // if(threadIdx.x==0 && blockIdx.x==0){
    // for(int i=0;i<count;i++){
    //     printf("i=%i, %f\n",i, A[i]);
    // }}
}




int main(int argc, char *argv[]){

    // hardware parameter for A3000
    // const int num_SM = 32;
    // const int num_warp = 32;
    // 4 warp schedulers.

    // input parameter
for(int test = 128; test<10000; test=test+32){

    int M = 10240;
    int N = 10240;
M=test;
N=test;

    //size_t num_thread = 512;
    //dim3 threadblock(warp, warp);
    //dim3 blockgrid( (unsigned int)(ceilf((float)M / (float)warp) ), (unsigned int)(ceilf((float)N / (float)warp) ) );
    
    float *A, *x, *y;
    // allocate memory on the device
    hipMalloc( &A, M*N * sizeof(float));
    hipMalloc( &x, N * sizeof(float));
    hipMalloc( &y, M * sizeof(float));

    // initialize variable value
    //cudaMemset is only for integer!!!
    //cudaMemset( A, 1, M*N);
    //cudaMemset( x, 3, N);
    //cudaMemset( y, 5, M);
    setmemoryf<<<(unsigned int)(ceilf((float)M*N / (float)num_thread) ), num_thread, 0, 0>>>(A, 1.0f, M*N);
    setmemoryf<<<(unsigned int)(ceilf((float)N / (float)num_thread) ), num_thread, 0, 0>>>(x, 3.0f, N);
    setmemoryf<<<(unsigned int)(ceilf((float)M / (float)num_thread) ), num_thread, 0, 0>>>(y, 0.0f, M);
    hipDeviceSynchronize();

    // compute and timing

        hipblasHandle_t handle;
        hipblasStatus_t stat = hipblasCreate(&handle);
        if (stat != HIPBLAS_STATUS_SUCCESS){
            printf("CUBLAS initialization failed\n");
            return 1;
        }

        const float alpha = 1.0f;
        const float beta  = 0.0f;




    int repeat = 1000;

    struct timespec start, end;
    timespec_get(&start, TIME_UTC);


    for(int i=0;i<repeat;i++){


        stat =hipblasSgemv(handle, HIPBLAS_OP_N, 
                          M, N, 
                          &alpha, A, M, 
                          x, 1, 
                          &beta, 
                          y, 1);
        hipDeviceSynchronize();


    }

    timespec_get(&end, TIME_UTC);

        if(stat != HIPBLAS_STATUS_SUCCESS){
            printf("CUBLAS operation failed\n");
            return 1;
        }
        
        hipblasDestroy(handle);






    time_t d_sec  = end.tv_sec  - start.tv_sec;
    long   d_nsec = end.tv_nsec - start.tv_nsec;
    double total_time = (double)d_sec + (double)d_nsec/1000000000.0;
    //printf("time=%f\n", total_time);
    double perf = 1.0e-9 * ((double)M*(double)N + (double)M + (double)N) * 4.0 / total_time * (double)repeat;
    printf("test= %i , time(sec)= %f ,memory throughput = %f GByte/s\n", test, total_time, perf);

    // show result
    //result<<<1,1>>>(y,M);


    // Free the memory on the device
    hipFree(A);
    hipFree(x);
    hipFree(y);
}
    return 0;
}